#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <time.h>
#include <iostream>
#include <fstream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

// export PROGRAM=time_matrix_multiplication_single.cu
// nvcc -c $PROGRAM.cu -o $PROGRAM.o && mpic++ -o $PROGRAM $PROGRAM.o -lcudart -L/usr/local/cuda/lib64 -I/usr/local/cuda/include
// mpirun -n1 ./$PROGRAM


using data_type = double;

__host__ std::vector<data_type> generateRandomMatrix(int size) {
    int totalElements = size * size;
    std::vector<data_type> matrix(totalElements);

    // Seed for random number generator
    //std::srand(statiResultcast<unsigned int>(std::time(nullptr)));

    // Initialize matrix with random values
    for (int i = 0; i < totalElements; ++i) {
        matrix[i] = static_cast<data_type>(std::rand() % 100); // Random integers (0-99)
    }

    return matrix;
}


__host__ void doRandomMatrixMutliplication(int start_size, int end_size, int increment, int iterations_per_size){
	hipblasHandle_t cublasH = NULL;
	hipStream_t stream = NULL;
	
	// begin printing our CSV data to stdout
	printf("Matrix multiplication\nMatrix Size, Execution Time(s), Number of Iterations\n");

	// begin timer
	clock_t start = clock();
	for (int size = start_size; size <= end_size; size = size+ increment){
		printf("%d",size);
		for (int i = 0; i <= iterations_per_size; i++){
			// create cuBLAS handle
			(hipblasCreate(&cublasH));

			(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
			(hipblasSetStream(cublasH, stream));
			hipblasHandle_t cublasH = NULL;
			hipStream_t stream = NULL;


			const std::vector<data_type> A = generateRandomMatrix(size);
			const std::vector<data_type> B = generateRandomMatrix(size);

			std::vector<data_type> C(size * size);
			const data_type alpha = 1.0;
			const data_type beta = 0.0;

			data_type *d_A = nullptr;
			data_type *d_B = nullptr;
			data_type *d_C = nullptr;

			hipblasOperation_t transa = HIPBLAS_OP_N;
			hipblasOperation_t transb = HIPBLAS_OP_N;


			/* step 1: create cublas handle, bind a stream */

			/* step 2: copy data to device */
			(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
			(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
			(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));

			(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
					       stream));
			(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
					       stream));

			/* step 3: compute */

			clock_t start = clock();
			(
			hipblasDgemm(cublasH, transa, transb, size, size, size, &alpha, d_A, size, d_B, size, &beta, d_C, size));

			/* step 4: copy data to host */
			(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,
					       stream));

			(hipStreamSynchronize(stream));
			
			


			/* free resources */
			(hipFree(d_A));
			(hipFree(d_B));
			(hipFree(d_C));
			(hipblasDestroy(cublasH));

			(hipStreamDestroy(stream));
		}
		clock_t end = clock();
		long double num_seconds = (long double)(end - start) / CLOCKS_PER_SEC;
		printf(",%Lf,%d\n",  num_seconds, iterations_per_size); 
	}
	(hipDeviceReset());

}



// __host__ void doRandomMatrixInversion(int size){
// 	hipblasHandle_t cublasH = NULL;
// 	hipStream_t stream = NULL;

// 	printf("Matrix inversion size: %d: ", size);


// 	const std::vector<data_type> A = generateRandomMatrix(size);
// 	const std::vector<data_type> Result = generateRandomMatrix(size);
// 	const std::vector<data_type> Info = generateRandomMatrix(size);

// 	std::vector<data_type> C(size * size);

// 	data_type *d_A = nullptr;
// 	data_type *d_Result = nullptr;
// 	data_type *d_Info = nullptr;

// 	/* step 1: create cublas handle, bind a stream */
// 	(hipblasCreate(&cublasH));

// 	(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
// 	(hipblasSetStream(cublasH, stream));

// 	/* step 2: copy data to device */
// 	(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
// 	(hipMalloc(reinterpret_cast<void **>(&d_Result), sizeof(data_type) * Result.size()));
// 	(hipMalloc(reinterpret_cast<void **>(&d_Info), sizeof(data_type) * Info.size()));

// 	(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
// 			       stream));
// 	(hipMemcpyAsync(d_Info, Info.data(), sizeof(data_type) * Info.size(), hipMemcpyHostToDevice,
// 			       stream));

// 	/* step 3: compute */
// 	//clock_t start = clock();
// 	(cublasDmatinvBatched(cublasH, 
// 		    size, 
// 		    (const double *const *) d_A, 
// 		    size, 
// 		    (double *const *)d_Result, 
// 		    size, 
// 		    (int *)d_Info, 
// 		    size));

// 	/* step 4: copy data to host */
// 	(hipMemcpyAsync((void *)Result.data(), d_Result, sizeof(data_type) * Result.size(), hipMemcpyDeviceToHost,
// 			       stream));

// 	/*
// 	hipStreamSynchronize(stream);
// 	clock_t end = clock();
// 	long double num_seconds = (long double)(end - start) / CLOCKS_PER_SEC;

// 	printf(" %Lf seconds\n",  num_seconds);    
// 	*/	    

// 	/* free resources */
// 	(hipFree(d_A));
// 	(hipFree(d_Info));
// 	hipFree(d_Result);
// 	//	    (hipFree(d_Result));

        

//         hipError_t err = hipGetLastError();
//         if (err != hipSuccess) {
//             fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
//             // Handle error appropriately
//         }
	            
// 	(hipblasDestroy(cublasH));

// 	(hipStreamDestroy(stream));

// 	(hipDeviceReset());
// }




int main(int argc, char** argv) {
	
	doRandomMatrixMutliplication(1500, 15000, 1500, 20);
    
    return 0;
}
